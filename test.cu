#include <hip/hip_runtime.h>
#include <stdio.h>
int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("Found %d CUDA-capable device(s)\n", deviceCount);
    return 0;
}